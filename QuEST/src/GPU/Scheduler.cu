#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#define INIT_LIST_SIZE 2048

enum func {CCU, HDM};

class List
{
private:
    int size;
    size_t typesize;
    void *begin;
    void *end;
    void *pitor;
    void addspace();
public:
    List(){};
    List(int ts);
    ~List();
    void reset();
    void* getbegin();
    size_t getdatacount();
    bool checktypesize(size_t n);
    template <typename T> void push(T* value, int count);
    template <typename T> void push(T* value);
    template <typename T> void itor(T &value);
    size_t copy(List* copylist);
};

class Scheduler
{
private:
    List* list4;
    List* list16;
    List* combinlist;
    int targetQubit;
    int funccount;
    size_t devicesize;
    void* device;
    void reset();
    template <typename T> List* listof();
public:
    Scheduler();
    ~Scheduler();
    template <typename T> void push(T value);
    template <typename T> void itor(T &value);
    void addfunc(Qureg qureg, const int newtargetQubit, func functype);
    void launch(Qureg qureg);
};

#ifdef __cplusplus
extern "C" {
#endif

__global__ void statevec_groupKernel(Qureg qureg, const int funccount, const int targetQubit, void* const list16, void* const list4);
//accept func
__global__ void statevec_controlledCompactUnitaryKernel (Qureg qureg, const int controlQubit, const int targetQubit, Complex alpha, Complex beta);
__global__ void statevec_hadamardKernel (Qureg qureg, const int targetQubit);

#ifdef __cplusplus
}
#endif


List::List(int ts){
    size = INIT_LIST_SIZE;
    typesize = ts;

    begin = malloc(size);
}

List::~List(){
    free(begin);
}

void List::addspace(){
    //malloc new space
    void *temp = malloc(size*2);
    //copy data
    memcpy(temp, begin, size);
    //change end pointer
    end = ((char*)end - (char*)begin) + (char*)temp;
    //free orignal space
    free(begin);
    //set new space
    begin = temp;
    //set new space size
    size <<= 1;
}

void List::reset(){
    end = begin;
    pitor = begin;
}

void* List::getbegin(){ return begin; }
size_t List::getdatacount(){ return (char*)end-(char*)begin; }
bool List::checktypesize(size_t n){ return (n == typesize); }

template <typename T>
void List::push(T* value, int count){
    size_t valuesize = sizeof(T) * count;
    //check if List is full
    while ( (char*)end + valuesize > (char*)begin + size ) addspace();
    //copy the data
    memcpy(end, value, valuesize);
    //move the end pointer
    end = (T*)end + count;
}

template <typename T>
void List::push(T* value){
    push<T>(value, 1);
}

template <typename T>
void List::itor(T &value){
    value = *((T*)pitor);
    pitor = (T*)pitor + 1;
}

size_t List::copy(List* copylist){
    size_t datacount = copylist->getdatacount();
    if (datacount) push<char>((char*)(copylist->getbegin()), datacount);
    return getdatacount();
}

Scheduler::Scheduler(){
    combinlist = new List(1);
    list4 = new List(4);
    list16 = new List(16);
    reset();
    devicesize = INIT_LIST_SIZE*2;
    if (hipSuccess != hipMalloc(&device, devicesize)) printf("cudamalloc failed!\n");
}

Scheduler::~Scheduler(){
    hipFree(device);
}

void Scheduler::reset(){
    funccount = 0;
    targetQubit = -1;
    list4->reset();
    list16->reset();
    combinlist->reset();
}

template <typename T>
List* Scheduler::listof(){
    switch (sizeof(T))
    {
    case 4:
        return list4;
    case 16:
        return list16;
    }
    return list4;
}

template <typename T>
void Scheduler::push(T value){
    List* list = listof<T>();

    list->push<T>(&value);
}

template <typename T>
void Scheduler::itor(T &value){
    List* list = listof<T>();

    list->itor<T>(value);    
}

void Scheduler::addfunc(Qureg qureg, const int newtargetQubit, func functype){
    if ((newtargetQubit != targetQubit) && (funccount != 0)) launch(qureg);
    targetQubit = newtargetQubit;
    push<func>(functype);
    ++funccount;
}

void Scheduler::launch(Qureg qureg){
    if (funccount==0) return;
    
    //re-launch
    if (funccount==1){
        func thisfunc;
        itor<func>(thisfunc);
        switch (thisfunc)
        {
        case CCU:{
            int threadsPerCUDABlock, CUDABlocks;
            threadsPerCUDABlock = 128;
            /* code */
            int controlQubit;
            Complex alpha,beta;
            itor<int>(controlQubit);
            itor<Complex>(alpha);
            itor<Complex>(beta);
            CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
            statevec_controlledCompactUnitaryKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, controlQubit, targetQubit, alpha, beta);
            break;
        }
        
        case HDM:{
            /* code */
            int threadsPerCUDABlock, CUDABlocks;
            threadsPerCUDABlock = 128;
            CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);
            statevec_hadamardKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, targetQubit);
            break;
        }
        
        default:
            break;
        }
    }
    else{
        int threadsPerCUDABlock, CUDABlocks;
        threadsPerCUDABlock = 512;
        CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk>>1)/threadsPerCUDABlock);

        size_t datacount16 = combinlist->copy(list16);
        size_t datacountall = combinlist->copy(list4);

        if ( datacountall > devicesize ){
            hipFree(device);
            devicesize = datacountall/16*16+16;
            if (hipSuccess != hipMalloc(&device, devicesize)) printf("cudamalloc failed!\n");
        }

        hipMemcpy(device, combinlist->getbegin(), datacountall, hipMemcpyHostToDevice);
        statevec_groupKernel<<<CUDABlocks, threadsPerCUDABlock>>>(qureg, funccount, targetQubit, device, (char*)device+datacount16);
    }

    reset();
}

template <typename T>
__forceinline__ __device__ void jump(void *&pointer, const int steplen){
    pointer = ((T*)pointer)+steplen;
}

template <typename T>
__forceinline__ __device__ void jump(void *&pointer){
    pointer = ((T*)pointer)+1;
}

template <typename T>
__forceinline__ __device__ void itor(void *&pointer, T &value){
    value = *((T*)pointer);
    jump<T>(pointer);
}